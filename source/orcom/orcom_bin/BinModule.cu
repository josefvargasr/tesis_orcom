#include "hip/hip_runtime.h"
/*
  This file is a part of ORCOM software distributed under GNU GPL 2 licence.
  Homepage:	http://sun.aei.polsl.pl/orcom
  Github:	http://github.com/lrog/orcom

  Authors: Sebastian Deorowicz, Szymon Grabowski and Lucas Roguski
*/

#include "Globals.h"

#include <vector>
#include <iostream>
#include <sys/time.h>

#include "BinModule.h"
#include "FastqStream.h"
#include "DnaParser.h"
#include "DnaPacker.h"
#include "DnaCategorizer.h"
#include "BinFile.h"
#include "BinOperator.h"
#include "DnaBlockData.h"
#include "Exception.h"
#include "Thread.h"

// CUDA
//
#ifdef CUDA
	//extern char* h_reads;
	extern uint32* arr_minim; 
#endif

extern uint32 countReads;
//clock_t start2, end1, start_1, end_2;
//extern clock_t end_1, start_2;

struct timeval start2, end1, start_1, end_2;
extern struct timeval end_1, start_2;
int diff_1, diff_2, diff_loop;

void BinModule::Fastq2Bin(const std::vector<std::string> &inFastqFiles_, const std::string &outBinFile_,
						  uint32 threadNum_,  bool compressedInput_, bool verboseMode_)
{
	// TODO: try/catch to free resources
	//
	IFastqStreamReader* fastqFile = NULL;
	if (compressedInput_)
		fastqFile = new MultiFastqFileReaderGz(inFastqFiles_);		// Abre el archivo Fastq Comprimido
	else
		fastqFile = new MultiFastqFileReader(inFastqFiles_);		// Abre el archivo Fastq sin Comprimir


	BinFileWriter binFile;
	binFile.StartCompress(outBinFile_, config);		// Prepara las configuraciones Iniciales del proceso de compresion

	const uint32 minimizersCount = config.minimizer.TotalMinimizersCount();
	if (threadNum_ > 1)
	{
		FastqChunkPool* fastqPool = NULL;
		FastqChunkQueue* fastqQueue = NULL;
		BinaryPartsPool* binPool = NULL;
		BinaryPartsQueue* binQueue = NULL;

		FastqChunkReader* fastqReader = NULL;
		BinChunkWriter* binWriter = NULL;

		const uint32 partNum = threadNum_ * 4;
		fastqPool = new FastqChunkPool(partNum, config.fastqBlockSize);
		fastqQueue = new FastqChunkQueue(partNum, 1);

		binPool = new BinaryPartsPool(partNum, minimizersCount);
		binQueue = new BinaryPartsQueue(partNum, threadNum_);

		fastqReader = new FastqChunkReader(fastqFile, fastqQueue, fastqPool);
		binWriter = new BinChunkWriter(&binFile, binQueue, binPool);

		// launch stuff
		//
		mt::thread readerThread(mt::ref(*fastqReader));

		std::vector<IOperator*> operators;
		operators.resize(threadNum_);

#ifdef USE_BOOST_THREAD
		boost::thread_group opThreadGroup;

		for (uint32 i = 0; i < threadNum_; ++i)
		{
			operators[i] = new BinEncoder(config.minimizer, config.catParams,
										  fastqQueue, fastqPool,
										  binQueue, binPool);
			opThreadGroup.create_thread(mt::ref(*operators[i]));
		}

		(*binWriter)();

		readerThread.join();
		opThreadGroup.join_all();


#else
		std::vector<mt::thread> opThreadGroup;

		for (uint32 i = 0; i < threadNum_; ++i)
		{
			operators[i] = new BinEncoder(config.minimizer, config.catParams,
										  fastqQueue, fastqPool, binQueue, binPool);
			opThreadGroup.push_back(mt::thread(mt::ref(*operators[i])));
		}

		(*binWriter)();

		readerThread.join();

		for (mt::thread& t : opThreadGroup)
		{
			t.join();
		}

#endif

		for (uint32 i = 0; i < threadNum_; ++i)
		{
			delete operators[i];
		}

		TFREE(binWriter);
		TFREE(fastqReader);

		TFREE(binQueue);
		TFREE(binPool);
		TFREE(fastqQueue);
		TFREE(fastqPool);
	}
	else
	{
		DnaParser parser;
		DnaCategorizer categorizer(config.minimizer, config.catParams);
		DnaPacker packer(config.minimizer);

		DataChunk fastqChunk(config.fastqBlockSize);
		std::vector<DnaRecord> records;										// Vector de objetos DnaRecords, que guarda los Reads leidos
		records.resize(1 << 10);
		//h_reads = (char*)malloc(10 * sizeof(char));
		DnaBinBlock dnaBins(minimizersCount);
		BinaryBinBlock binBins;
		DataChunk dnaBuffer;

#ifdef CUDA
		printf("\n----- Start  CUDA -----\n");
#else
		printf("\n----- Start ORCOM -----\n");
#endif				
		countReads=0;
		diff_1=0;
		diff_2=0;
		
		struct timeval start;
		gettimeofday(&start, NULL);
		gettimeofday(&end1, NULL);
		
		while (fastqFile->ReadNextChunk(&fastqChunk))						// Lee Partes del archivo Fastq
		{
			gettimeofday(&start_1, NULL);
			uint64 recordsCount = 0;
			parser.ParseFrom(fastqChunk, dnaBuffer, records, recordsCount);	// Toma las partes leidas y las pone en un vector de objetos, cada objeto contiene un Read
			
			ASSERT(recordsCount > 0);
			categorizer.Categorize(records, recordsCount, dnaBins);			// Busca los Minimizers y los almacena en Bins

			packer.PackToBins(dnaBins, binBins);							// Empaqueta los Bins

			binFile.WriteNextBlock(&binBins);								// Escribe el paquete de Bins
		
			gettimeofday(&end_2, NULL);
			diff_1 += (end_1.tv_sec - start_1.tv_sec);
			diff_2 += (end_2.tv_sec - start_2.tv_sec);
		}
		gettimeofday(&start2, NULL);
		struct timeval end;
		gettimeofday(&end, NULL);
		diff_loop = double(end.tv_sec - start.tv_sec);
		

		printf("Number of Minimizers processed: %d\n",countReads);
		printf("Number of Minimizers in file: %d\n",countReads/2);
#ifdef CUDA
		printf("\n----- End  CUDA -----\n");
#else
		printf("\n----- End ORCOM -----\n");
#endif
	}

	binFile.FinishCompress();

	if (verboseMode_)
	{
		std::vector<uint64> recordCounts;
		binFile.GetBinStats(recordCounts);

		std::cout << "Signatures count: " << recordCounts.size() << std::endl;
		std::cout << "Records distribution in bins by signature:\n";
		for (uint32 i = 0; i < recordCounts.size(); ++i)
		{
			if (recordCounts[i] > 0)
				std::cout << i << " : " << recordCounts[i] << '\n';
		}
		std::cout << std::endl;
	}

	delete fastqFile;
}


void BinModule::Bin2Dna(const std::string &inBinFile_, const std::string &outDnaFile_)
{
	// TODO: try/catch to free resources
	//
	BinFileReader binFile;

	binFile.StartDecompress(inBinFile_, config);
	uint32 minimizersCount = config.minimizer.TotalMinimizersCount();

	DnaFileWriter dnaFile(outDnaFile_);
	DataChunk fastqChunk(config.fastqBlockSize >> 1);			// WARNING! --- here can be a BUG
	DnaPacker packer(config.minimizer);
	DnaParser parser;

	DnaBinBlock dnaBins(minimizersCount);
	BinaryBinBlock binBins;
	DataChunk dnaBuffer;

	while (binFile.ReadNextBlock(&binBins))
	{
		packer.UnpackFromBins(binBins, dnaBins, dnaBuffer);
		parser.ParseTo(dnaBins, fastqChunk);

		dnaFile.WriteNextChunk(&fastqChunk);
	}

	dnaFile.Close();
	binFile.FinishDecompress();
}
