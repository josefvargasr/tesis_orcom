#include "hip/hip_runtime.h"
/*
  This file is a part of ORCOM software distributed under GNU GPL 2 licence.
  Homepage:	http://sun.aei.polsl.pl/orcom
  Github:	http://github.com/lrog/orcom

  Authors: Sebastian Deorowicz, Szymon Grabowski and Lucas Roguski
*/

#include "Globals.h"

#include <string.h>
#include <algorithm>
#include <sys/time.h>

#include "DnaCategorizer.h"
#include "DnaBlockData.h"

// CUDA
// -----------------------------------
uint32 countReads = 0;

double total_f1 = 0, total_f2 = 0, total_f3 = 0, diff_f1 = 0, diff_f2 = 0, diff_f3 = 0;
uint32 countCategorize = 0;

struct timeval start_2, end_1, start_bin, start_bin2, end_bin, end_bin2;
int diff_bin, diff_bin2, diff_f33, diff__f1 = 0, total_f33 = 0;

#ifdef CUDA
	__device__ char d_symbolIdxTable[128];
	__device__ uint32 d_nBinValue;
	__device__ uint32 d_maxLongMinimValue;
	__device__ uint32 d_maxShortMinimValue;			
	__device__ uint8 d_params_signatureLen;
	__device__ uint8 d_params_skipZoneLen; 
	uint64 h_dnaSize;	
	char* h_reads;
	char* d_reads;char* d_readsRC;
	uint32* h_posReads;
	uint32* d_posReads;
	uint16* h_lenReads;
	uint16* d_lenReads;
	uint32* h_arr_minim;
	uint32* d_arr_minim;
	#define numStream	8	
	#define STREAMS		1	
#endif
// -----------------------------------

DnaCategorizer::DnaCategorizer(const MinimizerParameters& params_, const CategorizerParameters& catParams_)
	:	params(params_)
	,	catParams(catParams_)
	,	maxShortMinimValue(1 << (2 * params.signatureSuffixLen))
	,	maxLongMinimValue(1 << (2 * params.signatureLen))
	,	nBinValue(maxLongMinimValue)
{
	ASSERT(params.signatureSuffixLen <= params.signatureLen);

	std::fill(symbolIdxTable, symbolIdxTable + 128, -1);		// Llena el vector symbolIdxTable con '-1'
	for (uint32 i = 0; i < 5; ++i)
		symbolIdxTable[(int32)params.dnaSymbolOrder[i]] = i;	// En la posicion de cada letra (ACGTN) poner su respectivo Id
																// A = symbolIdxTable[65] = 0; C = symbolIdxTable[67] = 1; G = symbolIdxTable[71] = 2;
																// T = symbolIdxTable[84] = 3; N = symbolIdxTable[78] = 4;
	freqTable.resize(maxShortMinimValue, 0);
}


void DnaCategorizer::Categorize(std::vector<DnaRecord>& records_, uint64 recordsCount_, DnaBinBlock& bin_)
{
	
	ASSERT(recordsCount_ > 0);
	ASSERT(recordsCount_ <= records_.size());

	// clear bins
	//
	for (uint32 i = 0; i < bin_.stdBins.Size(); ++i)
		bin_.stdBins[i].Clear();
	bin_.nBin.Clear();

	std::fill(freqTable.begin(), freqTable.end(), 0);

	// process records
	//
	countCategorize++;
	
	clock_t start_f1 = clock();

	gettimeofday(&end_1, NULL);
	DistributeToBins(records_, recordsCount_, bin_.stdBins, bin_.nBin);		// Lee cada Read del vector records_, encuentra su minimizador y guarda el Read en el Bin correspondiente

	gettimeofday(&start_2, NULL);

	clock_t end_f1 = clock();	
	diff_f1 = 0;
	diff_f1 = double(end_f1 - start_f1)/CLOCKS_PER_SEC;
	total_f1 += diff_f1;

	// sort the bins
	//

	FindMinimizerPositions(bin_.stdBins);
	
	
	DnaRecordComparator comparator(params.signatureLen - params.signatureSuffixLen);


	for (uint32 i = 0; i < bin_.stdBins.Size(); ++i)
	{
		DnaBin& db = bin_.stdBins[i];
		if (db.Size() > 0)
			std::sort(db.Begin(), db.End(), comparator);
	}
		
	
	printf("F1 Time: %fs, Total: %fs ---- DistributeToBins\n", diff_f1, total_f1);
	printf("F2 Time: %fs, Total: %fs ---- FindMinimizer\n", diff_f2, total_f2);
	printf("F3 Time: %fs, Total: %fs ---- PrepReads, countCategorize: %d\n", diff_f3, total_f3, countCategorize);	
	printf("F33 Time: %ds, Total: %ds ---- GPU\n", total_f33, diff_f33);

}

#ifdef CUDA

#define HANDLE_ERROR(err) handleError(err, __FILE__, __LINE__)

static void handleError(hipError_t err, const char *file, int line){
	if(err != hipSuccess){
		printf("Cuda error at %s:%d: %s\n", file, line, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void getCudaMemInfo(){
	size_t free_byte, total_byte;

	hipMemGetInfo(&free_byte, &total_byte);

	double free = (double) free_byte;
	double total = (double) total_byte;
	double used = total - free;

	printf("Cuda Memory. Used: %fMB, Free: %fMB, Total: %fMB\n", used/1024/1024, free/1024/1024, total/1024/1024);
}

#define N_THREADS	512//256//512//1024
#define N_BLOCKS	32767//16384//32767//65535

__device__ bool d_IsMinimizerValid(uint32_t minim_, uint32_t mLen_)
{
        if(minim_ == 0)
                minim_ = 0;

        const uint32_t excludeLen = 3;
        const uint32_t excludeMask = 0x3F;
        const uint32_t symbolExcludeTable[] = {0x00, 0x15, 0x2A, 0x3F};

        minim_ &= (1 << (2 * mLen_)) - 1;
        bool hasInvalidSeq = false;
        for (uint32_t i = 0; !hasInvalidSeq && i <= (mLen_ - excludeLen); ++i){
                uint32_t x = minim_ & excludeMask;

                for(uint32_t j = 0; j < 4; ++j){
                        hasInvalidSeq |= (x == symbolExcludeTable[j]);
                }

                minim_ >>= 2;
        }

        return !hasInvalidSeq;
}

__device__ uint32 d_ComputeMinimizer(const char* dna_, uint32_t mLen_)
{
        uint32_t r = 0;

        for(uint32_t i = 0; i < mLen_; ++i){

                if(dna_[i] == 'N')
                        return d_nBinValue;

                r <<= 2;
                r |= d_symbolIdxTable[(uint32_t)dna_[i]];
        }

        return r;
}

__device__ uint32 d_FindMinimizer(char * d_dna_, uint16 d_len_)
{
	uint32 minimizer = d_maxLongMinimValue;
	#if EXP_USE_RC_ADV
		const int32 ibeg = 0; //rec_.reverse ? d_params_skipZoneLen : 0;
		const int32 iend = d_len_ - d_params_signatureLen + 1 - (rec_.reverse ? 0 : d_params_skipZoneLen);
	#else	
		const int32 ibeg = 0;
		const int32 iend = d_len_ - d_params_signatureLen + 1 - d_params_skipZoneLen;
	#endif
	for (int32 i = ibeg; i < iend; ++i){
		uint32 m = d_ComputeMinimizer(d_dna_ + i, d_params_signatureLen);		
		if (m < minimizer && d_IsMinimizerValid(m, d_params_signatureLen))		
			minimizer = m;
	}
	
	if (minimizer >= d_maxLongMinimValue)
		return d_nBinValue;

	return minimizer & (d_maxShortMinimValue - 1);
}

__device__ char* d_ComputeRC(char* d_reads_, char* d_readsRC_, uint16 d_len_)
{
	const char rcCodes[24] = {-1,'T',-1,'G',-1,-1,-1,'C',
				  -1,-1,-1,-1,-1,-1,'N',-1,
				  -1,-1,-1,-1,'A',-1,-1,-1,
				};
		
	for(uint32 i = 0; i < d_len_; ++i){
		d_readsRC_[d_len_-1-i] = rcCodes[(int32)d_reads_[i] - 64];
	}
	
	return d_readsRC_;
}

#ifndef STREAMS
__global__ void d_DistributeToBins(uint32 n_reads_, uint32* d_arr_minim_, uint16* d_lenReads_, uint32* d_posReads_, char* d_reads, char* d_readsRC)
#else
__global__ void d_DistributeToBins(uint32 n_reads_, uint32* d_arr_minim_, uint16* d_lenReads_, uint32* d_posReads_, char* d_reads, char* d_readsRC, uint32 offset, uint32 sizeOffset2)
#endif
{
#ifdef STREAMS
	uint32 tid = offset +  threadIdx.x + (blockDim.x * blockIdx.x);
	uint32 tid2 = threadIdx.x + (blockDim.x * blockIdx.x);
	if(tid < n_reads_ && tid2 < sizeOffset2 )
#else
	uint32 tid = threadIdx.x + (blockDim.x * blockIdx.x);
	if(tid < n_reads_ )
#endif
		
	{
		d_arr_minim_[tid] = d_FindMinimizer(&d_reads[d_posReads_[tid]], d_lenReads_[tid]);
		d_arr_minim_[tid + n_reads_] = d_FindMinimizer(d_ComputeRC(&d_reads[d_posReads_[tid]], &d_readsRC[d_posReads_[tid]], d_lenReads_[tid]), d_lenReads_[tid]);
	}
}

#endif

// todo: split into rev and non-rev
void DnaCategorizer::DistributeToBins(std::vector<DnaRecord>& records_, uint64 recordsCount_, DnaBinCollection& bins_, DnaBin& nBin_)
{	
	struct timeval start__f1;
	gettimeofday(&start__f1, NULL);

	char revBuffer[1024];		// TODO: make size constant depending on the record max len
	DnaRecord rcRec;		//Objeto que guardar el complemento reverso del Read
	rcRec.dna = revBuffer;
	rcRec.reverse = true;		//Indica que el objeto guarda el Read de forma inversa
	
#ifdef CUDA
	uint32 n_reads = 0;
	uint16 h_len_ = records_[0].len;
	if(countReads == 0){
		printf("Longitud del primer Read: %d\n", h_len_);
		// Alojar variables globales en GPU
		uint32* dd_maxLongMinimValue;
		HANDLE_ERROR(hipGetSymbolAddress((void**)&dd_maxLongMinimValue, d_maxLongMinimValue));
		HANDLE_ERROR(hipMemcpy(dd_maxLongMinimValue, &maxLongMinimValue, sizeof(uint32), hipMemcpyHostToDevice));
		uint32* dd_maxShortMinimValue;
		HANDLE_ERROR(hipGetSymbolAddress((void**)&dd_maxShortMinimValue, d_maxShortMinimValue));
		HANDLE_ERROR(hipMemcpy(dd_maxShortMinimValue, &maxShortMinimValue, sizeof(uint32), hipMemcpyHostToDevice));
		uint8* dd_params_signatureLen;
		HANDLE_ERROR(hipGetSymbolAddress((void**)&dd_params_signatureLen, d_params_signatureLen));
		HANDLE_ERROR(hipMemcpy(dd_params_signatureLen, &params.signatureLen, sizeof(uint8), hipMemcpyHostToDevice));
		uint8* dd_params_skipZoneLen;
		HANDLE_ERROR(hipGetSymbolAddress((void**)&dd_params_skipZoneLen, d_params_skipZoneLen));
		HANDLE_ERROR(hipMemcpy(dd_params_skipZoneLen, &params.skipZoneLen, sizeof(uint8), hipMemcpyHostToDevice));
		char* dd_symbolIdxTable;
		HANDLE_ERROR(hipGetSymbolAddress((void**)&dd_symbolIdxTable, d_symbolIdxTable));
		HANDLE_ERROR(hipMemcpy(dd_symbolIdxTable, symbolIdxTable, 128 * sizeof(char), hipMemcpyHostToDevice));
		uint32* dd_nBinValue;
		HANDLE_ERROR(hipGetSymbolAddress((void**)&dd_nBinValue, d_nBinValue));
		HANDLE_ERROR(hipMemcpy(dd_nBinValue, &nBinValue, sizeof(uint32), hipMemcpyHostToDevice));
		
		HANDLE_ERROR(hipMalloc((void**)&d_readsRC, h_dnaSize * sizeof(char)));
		HANDLE_ERROR(hipMalloc((void**)&d_reads, h_dnaSize * sizeof(char)));
		HANDLE_ERROR(hipMalloc((void**)&d_lenReads, recordsCount_ * sizeof(uint16)));
		h_arr_minim = (uint32*)malloc(2 * recordsCount_ * sizeof(uint32));
		h_posReads = (uint32*)malloc(recordsCount_ * sizeof(uint32));
		HANDLE_ERROR(hipMalloc((void**)&d_posReads, recordsCount_ * sizeof(uint32)));
		HANDLE_ERROR(hipMalloc((void**)&d_arr_minim, 2 * recordsCount_ * sizeof(uint32)));
	}	
#endif
	
	struct timeval end__f1;
	gettimeofday(&end__f1, NULL);
	diff__f1 = (end__f1.tv_sec - start__f1.tv_sec);

	if (params.tryReverseCompliment)					// Si se usa el Read en forma reversa
	{
	#ifdef CUDA
		clock_t start_f3 = clock();

		uint32 pos = 0;
		for(uint32 i = 0; i < recordsCount_; ++i){
			h_posReads[i] = pos;
			pos += records_[i].len;
			h_lenReads[i] = records_[i].len;
		}

		n_reads = recordsCount_;
		countReads += n_reads * 2;
		clock_t end_f3 = clock();	
		diff_f3 = 0;
		diff_f3 = double(end_f3 - start_f3)/CLOCKS_PER_SEC;
		total_f3 += diff_f3;

		struct timeval start_f33;
		gettimeofday(&start_f33, NULL);

	#ifndef STREAMS

		// Copiar Datos a la GPU
		HANDLE_ERROR(hipMemcpy(d_reads, h_reads, h_dnaSize * sizeof(char), hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(d_lenReads, h_lenReads, recordsCount_ * sizeof(uint16), hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(d_posReads, h_posReads, recordsCount_ * sizeof(uint32), hipMemcpyHostToDevice));

		// Lanzar Kernel
		d_DistributeToBins<<<((n_reads) + N_THREADS-1)/N_THREADS, N_THREADS>>>(n_reads, d_arr_minim, d_lenReads, d_posReads, d_reads, d_readsRC);
		
		HANDLE_ERROR(hipGetLastError());
		HANDLE_ERROR(hipDeviceSynchronize());
		//getCudaMemInfo();
	
		// Copiar Datos a la CPU
		HANDLE_ERROR(hipMemcpy(h_arr_minim, d_arr_minim, 2 * recordsCount_ * sizeof(uint32), hipMemcpyDeviceToHost));

	#else

		const int n = (n_reads) + (N_THREADS-1);
		const int nStreams = numStream;

		// Crear Streams
		hipStream_t stream[numStream];
		for (uint32 i = 0; i < nStreams; ++i)
			hipStreamCreate(&stream[i]);
		
		// Copiar Datos a la GPU y Lanzar Kernel por cada Stream
		for (uint32 i = 0; i < nStreams; ++i)
		{	
			uint32 offset1 = i * (h_dnaSize/nStreams);
			uint32 offset2 = i * (recordsCount_/nStreams);
			uint32 sizeOffset1;
			uint32 sizeOffset2;
			uint32 bSize = (((n)/N_THREADS)/nStreams) + 1;

			if(i == nStreams - 1){
				sizeOffset1 = h_dnaSize/nStreams;
				sizeOffset1 = sizeOffset1 + (h_dnaSize - (sizeOffset1 * nStreams));
				sizeOffset2 = recordsCount_/nStreams;
				sizeOffset2 = sizeOffset2 + (recordsCount_ - (sizeOffset2 * nStreams));
			}else{
				sizeOffset1 = h_dnaSize/nStreams;
				sizeOffset2 = recordsCount_/nStreams;
			}

			HANDLE_ERROR(hipMemcpyAsync(&d_reads[offset1], &h_reads[offset1], sizeOffset1 * sizeof(char), hipMemcpyHostToDevice, stream[i]));
			HANDLE_ERROR(hipMemcpyAsync(&d_lenReads[offset2], &h_lenReads[offset2], sizeOffset2 * sizeof(uint16), hipMemcpyHostToDevice, stream[i]));
			HANDLE_ERROR(hipMemcpyAsync(&d_posReads[offset2], &h_posReads[offset2], sizeOffset2 * sizeof(uint32), hipMemcpyHostToDevice, stream[i]));
			d_DistributeToBins<<< bSize, N_THREADS, 0, stream[i]>>>(n_reads, d_arr_minim, d_lenReads, d_posReads, d_reads, d_readsRC, offset2, sizeOffset2);
		}	
		
		for (uint32 i = 0; i < nStreams; ++i)
			HANDLE_ERROR(hipStreamSynchronize(stream[i]));
		
		// Copiar Datos a la CPU
		for (uint32 i = 0; i < nStreams; ++i)	
		{
			uint32 sizeOffset2;
			uint32 offset2 = i * 2 * (recordsCount_/nStreams);

			if(i == nStreams - 1){
				sizeOffset2 = recordsCount_/nStreams;
				sizeOffset2 = sizeOffset2 + (recordsCount_ - (sizeOffset2 * nStreams));
			}else{
				sizeOffset2 = recordsCount_/nStreams;
			}	
			HANDLE_ERROR(hipMemcpyAsync(&h_arr_minim[offset2], &d_arr_minim[offset2], 2 * sizeOffset2 * sizeof(uint32), hipMemcpyDeviceToHost, stream[i]));
		}

		
		// Destruir Streams
		for (uint32 i = 0; i < nStreams; ++i)
			hipStreamDestroy(stream[i]);
		
		
	#endif
		struct timeval end_f33;
		gettimeofday(&end_f33, NULL);

		total_f33 = (end_f33.tv_sec - start_f33.tv_sec);
		diff_f33 += total_f33;

		gettimeofday(&start_bin, NULL);
				
		for (uint32 i = 0; i < recordsCount_; ++i)
		{
			DnaRecord& rec = records_[i];				//Objeto que guardar el Read de forma directa
			rec.reverse = false;
	
			rec.ComputeRC(rcRec);
			
			const uint32 minimizerFwd = h_arr_minim[(i)];			
			const uint32 minimizerRev = h_arr_minim[(i) + (recordsCount_)];
			uint32 minimizer = 0;
			bool reverse = false;
		
			if (minimizerFwd <= minimizerRev)			
			{
				minimizer = minimizerFwd;
			}
			else
			{
				minimizer = minimizerRev;
				reverse = true;
			}

			// store record to bin
			//
			if (minimizer != nBinValue)				// !TODO --- find here minimizer pos
			{
				if (reverse)
				{
					rec.reverse = true;
					std::copy(rcRec.dna, rcRec.dna + rec.len, rec.dna);
				}

				bins_[minimizer].Insert(rec);			// Guarda el read en el Bin corresspondiente dependiedo del minimizador
			}
			else
			{
				rec.reverse = false;
				rec.minimizerPos = 0;
				nBin_.Insert(rec);				// Guarda el read en el Bin corresspondiente si no tiene minimizador
			}
		}
		
		gettimeofday(&end_bin, NULL);
  		int diffa = (end_bin.tv_sec - start_bin.tv_sec);
		diff_bin += diffa;
	
	#else
		
		
		for (uint32 i = 0; i < recordsCount_; ++i){
			DnaRecord& rec = records_[i];				//Objeto que guardar el Read de forma directa
			rec.reverse = false;

			ASSERT(rec.len > 0);

			rec.ComputeRC(rcRec);					//Se encarga de buscar el complemento inverso del Read

			// find and select minimizers
			//
			const uint32 minimizerFwd = FindMinimizer(rec);		// Busca el minimizador del read en forma directa
			const uint32 minimizerRev = FindMinimizer(rcRec);	// Busca el minimizador del complemento inverso del Read

			uint32 minimizer = 0;
			bool reverse = false;

			if (minimizerFwd <= minimizerRev)			// Verifica si elige minimizerFwd o minimizerRev
			{
				minimizer = minimizerFwd;
			}
			else
			{
				minimizer = minimizerRev;
				reverse = true;
			}

			// store record to bin
			//
			if (minimizer != nBinValue)				// !TODO --- find here minimizer pos
			{
				if (reverse)
				{
					rec.reverse = true;
					std::copy(rcRec.dna, rcRec.dna + rec.len, rec.dna);
				}

				bins_[minimizer].Insert(rec);			// Guarda el read en el Bin corresspondiente dependiedo del minimizador
			}
			else
			{
				rec.reverse = false;
				rec.minimizerPos = 0;
				nBin_.Insert(rec);				// Guarda el read en el Bin corresspondiente si no tiene minimizador
			}
		}
	#endif

	}
	else
	{
		printf("NO tryReverseCompliment\n");
		for (uint32 i = 0; i < recordsCount_; ++i)
		{
			DnaRecord& r = records_[i];
			ASSERT(r.len > 0);
			ASSERT(!r.reverse);
			uint32 minimizer = FindMinimizer(r);			// Encuentra el minimizador

			if (minimizer != nBinValue)				// !TODO --- find here minimizer pos
			{
				bins_[minimizer].Insert(r);			// Guarda el read en el Bin corresspondiente dependiedo del minimizador
			}
			else
			{
				r.minimizerPos = 0;
				nBin_.Insert(r);				// Guarda el read en el Bin corresspondiente si no tiene minimizador
			}
		}
	}

	// re-balance bins
	//
	gettimeofday(&start_bin2, NULL);

	for (uint32 i = 0; i < bins_.Size(); ++i)
	{
		DnaBin& db = bins_[i];
		if (db.Size() == 0 || db.Size() >= catParams.minBlockBinSize)
			continue;

		for (uint32 j = 0; j < db.Size(); ++j)
		{
			DnaRecord& r = db[j];
			r.minimizerPos = 0;

			// un-reverse the record
			if (r.reverse)
			{
				r.ComputeRC(rcRec);
				r.reverse = false;
				std::copy(rcRec.dna, rcRec.dna + r.len, r.dna);
			}

			std::map<uint32, uint16> mins = FindMinimizers(r);
			std::map<uint32, uint16>::iterator mit = mins.begin();

			for ( ; mit != mins.end(); ++mit)
			{
				const uint32 m = mit->first;
				if (bins_[m].Size() >= catParams.minBlockBinSize)// && bins_[m].Size() < maxBinSize)
				{
					r.minimizerPos = mit->second;
					bins_[m].Insert(r);
					break;
				}
			}

			// only one minimizer or we did not find appropriate bin
			if (mit == mins.end())
			{
				nBin_.Insert(r);
			}
		}
		db.Clear();
	}

	gettimeofday(&end_bin2, NULL);
	int diffb = (end_bin2.tv_sec - start_bin2.tv_sec);
	diff_bin2 += diffb;
		
}


void DnaCategorizer::FindMinimizerPositions(DnaBinCollection& bins_)
{
	
	for (uint32 i = 0; i < params.TotalMinimizersCount(); ++i)
	{
		if (bins_[i].Size() == 0)
			continue;

		char minString[64] = {0};
		params.GenerateMinimizer(i, minString);

		for (uint32 j = 0; j < bins_[i].Size(); ++j)
		{
			DnaRecord& r = bins_[i][j];
	#if EXP_USE_RC_ADV
			const char* beg = r.dna + (r.reverse ? params.skipZoneLen : 0);
			const char* end = r.dna + r.len - (r.reverse ? params.skipZoneLen : 0);
			const char* mi = std::search(beg, end, minString, minString + params.signatureSuffixLen);

			ASSERT(mi != r.dna + r.len);

			r.minimizerPos = mi - r.dna;
			ASSERT((!r.reverse && r.minimizerPos < r.len - params.skipZoneLen) ||
					(r.minimizerPos >= params.skipZoneLen) );
	#else
			const char* mi = std::search(r.dna, r.dna + r.len, minString, minString + params.signatureSuffixLen);

			ASSERT(mi != r.dna + r.len);

			r.minimizerPos = mi - r.dna;
			ASSERT(r.minimizerPos < r.len - params.skipZoneLen);
	#endif

		}
	}
}

uint32 DnaCategorizer::FindMinimizer(DnaRecord &rec_)
{
	
	uint32 minimizer = maxLongMinimValue;
	ASSERT(rec_.len >= params.signatureLen - params.skipZoneLen + 1);
	
	// Calculo de la posicion donde se comienza a iterar y numero de iteraciones que se realizarn en el Read para encontrar el minimizer
	#if EXP_USE_RC_ADV
		const int32 ibeg = rec_.reverse ? params.skipZoneLen : 0;
		const int32 iend = rec_.len - params.signatureLen + 1 - (rec_.reverse ? 0 : params.skipZoneLen);
	#else
		const int32 ibeg = 0;
		const int32 iend = rec_.len - params.signatureLen + 1 - params.skipZoneLen;
	#endif

	for (int32 i = ibeg; i < iend; ++i){
		uint32 m = ComputeMinimizer(rec_.dna + i, params.signatureLen);		
		if (m < minimizer && IsMinimizerValid(m, params.signatureLen))		
			minimizer = m;
	}

	countReads++;

	//printf("%d, Minimizer: %d\n", countReads, minimizer);
	
	if (minimizer >= maxLongMinimValue)
		return nBinValue;

	return minimizer & (maxShortMinimValue - 1);
}

std::map<uint32, uint16> DnaCategorizer::FindMinimizers(DnaRecord &rec_)
{
	ASSERT(rec_.len >= params.signatureLen - params.skipZoneLen + 1);

	// find all
	std::map<uint32, uint16> signatures;
	for (int32 i = 0; i < rec_.len - params.signatureLen + 1 - params.skipZoneLen; ++i)
	{
		uint32 m = ComputeMinimizer(rec_.dna + i, params.signatureLen);

		if (IsMinimizerValid(m, params.signatureLen) && m < maxLongMinimValue)
		{
			m &= (maxShortMinimValue - 1);
			if (signatures.count(m) == 0)
				signatures[m] = i + (params.signatureLen - params.signatureSuffixLen);
		}
	}

	return signatures;
}


uint32 DnaCategorizer::ComputeMinimizer(const char* dna_, uint32 mLen_)
{
	uint32 r = 0;						// Minimizador Retornado en binario

	for (uint32 i = 0; i < mLen_; ++i)			// Recorre toda la cadena de caracteres
	{
		if (dna_[i] == 'N')				// Si alguna letra es 'N' retorna un valor predeterminado
			return nBinValue;

		ASSERT(dna_[i] >= 'A' && dna_[i] <= 'T');	// Si el caracter es correcto continua 
		r <<= 2;					// Desplaza 'r' dos bits a la izquierda
		r |= symbolIdxTable[(uint32)dna_[i]];		// Hace r = r OR symbolIdxTable[ACGT]
	}

	return r;						// Variable que contiene el minimizador hallado en binario
}


bool DnaCategorizer::IsMinimizerValid(uint32 minim_, uint32 mLen_)
{
	if (minim_ == 0)
		minim_ = 0;

	const uint32 excludeLen = 3;
	const uint32 excludeMask = 0x3F;
	const uint32 symbolExcludeTable[] = {0x00, 0x15, 0x2A, 0x3F};	//0, 21, 42, 63

	minim_ &= (1 << (2*mLen_)) - 1;
	bool hasInvalidSeq = false;

	for (uint32 i = 0; !hasInvalidSeq && i <= (mLen_ - excludeLen); ++i)
	{
		uint32 x = minim_ & excludeMask;

		for (uint32 j = 0; j < 4; ++j)
			hasInvalidSeq |= (x == symbolExcludeTable[j]);

		minim_ >>= 2;
	}

	return !hasInvalidSeq;
}
